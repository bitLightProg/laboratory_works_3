#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <ctime>
#include <fstream>
#include <Windows.h>

__device__ unsigned int block_id = -1;


__global__ void div(float* arr, int n, int position, float* p) {
	__shared__ float beg;
	beg = arr[(n)*(blockIdx.x + position) + position];
	if (threadIdx.x == 0) {
		beg = arr[(n)*(blockIdx.x + position) + position];

		int current_index = blockIdx.x;
		if (beg == 0 && current_index != 0)
			return;
		while (block_id != current_index - 1);
		*p *= beg;
		++block_id;
	}

	unsigned int x = threadIdx.x + position;
	unsigned int y = blockIdx.x + position;
	
	if ((x >= n) || (y >= n)) {
		return;
	}
	
	if (beg != 0) {
		arr[y*(n)+x] /= beg;
	}
}
__global__ void sub(float* arr, int n, int position) {
	block_id = -1;
	__shared__ float beg;
	beg = 0;
	if (threadIdx.x == 0) {
		beg = arr[(n)*(blockIdx.x + position) + position];
	}
	
	unsigned int x = threadIdx.x + position;
	unsigned int y = blockIdx.x + position;
	__syncthreads();
	if ((x >= n) || (y >= n)) {
		return;
	}
	if (beg == 0) {
		return;
	}
	arr[y*(n)+x] -= arr[n*position + x];

}

void read_from_file(std::ifstream& input, int& n, float*& arr) {
	input.read((char*)&n, sizeof(int));
	arr = new float[n*n];
	input.read((char*)arr, n*n * sizeof(float));
}

void write_to_file(std::ostream &os, float*& arr, int& n) {
	os.write((char*)&n, sizeof(int));
	os.write((char*)arr, n*n * sizeof(float));
}

void rutine(float*& arr, int& n, int& position, float& p) {
	for (int i = position; i < n; ++i) {
		float beg = arr[i*n + position];
		if (beg == 0 && i != position)
			continue;
		p *= beg;

		for (int j = position; j < n; ++j) {
			arr[i*n + j] /= beg;
		}
	}

	for (int i = position + 1; i < n; ++i) {
		if (arr[i*n + position] == 0) {
			continue;
		}
		for (int j = position; j < n; ++j) {
			arr[i*n + j] -= arr[position*n + j];
		}
	}

	++position;
}

int main() {
	SetConsoleCP(1251);
	SetConsoleOutputCP(1251);
	std::cout << "1. Ввести с клавиатуры." << std::endl;
	std::cout << "2. Читать из бинарного файла." << std::endl;
	std::cout << "3. Комплексное тестирование." << std::endl;
	int chose;
	std::cin >> chose;
	int n;
	float *arr = nullptr;
	int position = 0;
	float p = 1;

	if (chose == 1) {
		std::cout << "Введите размерность матрицы." << std::endl;
		
		std::cin >> n;
		std::cout << "Введите элементы матрицы." << std::endl;
		srand(time(NULL));
		arr = new float[n*n];
		for (int i = 0; i < n; ++i) {
			for (int j = 0; j < n; ++j) {
				std::cin >> arr[i*n + j];
			}
		}
		std::cout << std::endl;
	}
	else if (chose == 2) {
		std::ifstream in_bin("input.bin", std::ios_base::binary);
		read_from_file(in_bin, n, arr);
	}
	else if (chose == 3) {
		std::cout << "Введите размерность матрицы." << std::endl;
		std::cin >> n;
		arr = new float[n*n];
		srand(time(NULL));
		for (int i = 0; i < n; ++i) {
			for (int j = 0; j < n; ++j) {
				arr[i*n + j] = 1 + rand() % 10;
				rand() % 2 ? arr[i*n + j] : arr[i*n + j] = -arr[i*n + j];
			}
		}
		std::ofstream out("input.bin", std::ios_base::binary);
		write_to_file(out, arr, n);
		SYSTEMTIME s1, s2;
		GetLocalTime(&s1);
		while (position != n) {
			rutine(arr, n, position, p);
		}
		GetLocalTime(&s2);
		std::cout << "Определитель: " << p << std::endl;
		std::cout << "Время: " << (((s2.wHour - s1.wHour)*60 + s2.wMinute - s1.wMinute) * 60 + s2.wSecond - s1.wSecond) * 1000
			+ s2.wMilliseconds - s1.wMilliseconds << std::endl;
	}
	else return 0;

	float* dev_arr = nullptr;
	float* dev_p = nullptr;
	int size = n*n;
	position = 0;
	float k = p;
	p = 1;
	if (chose == 3) {
		std::ifstream in_bin("input.bin", std::ios_base::binary);
		read_from_file(in_bin, n, arr);
		in_bin.close();
	}	

	hipMalloc((void**)&dev_arr, size * sizeof(float));
	hipMalloc((void**)&dev_p, sizeof(float));

	hipMemcpy(dev_p, &p, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr, arr, size * sizeof(float), hipMemcpyHostToDevice);

	SYSTEMTIME s1, s2;
	GetLocalTime(&s1);
	while (position != n) {
		div<<<n-position, n - position >>>(dev_arr, n, position, dev_p);		
		sub<<<n - position, n - position >>>(dev_arr, n, position);
		
		++position;
	}
	hipMemcpy(&p, dev_p, sizeof(float), hipMemcpyDeviceToHost);
	GetLocalTime(&s2);
	
	std::cout << "Определитель: " << p << std::endl;
	std::cout << "Время: " << ((s2.wMinute - s1.wMinute) * 60 + s2.wSecond - s1.wSecond) * 1000
		+ s2.wMilliseconds - s1.wMilliseconds << std::endl;
	system("pause");
	hipFree(dev_arr);
	hipFree(dev_p);
	delete[] arr;
	return 0;
}

