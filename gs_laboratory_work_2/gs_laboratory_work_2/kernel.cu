#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <ctime>
#include <Windows.h>

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//	system("pause");
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}

void print(float* arr, int n) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			std::cout << arr[i*n + j] << " ";
		}
		std::cout << std::endl;
	}
	/*std::cout << std::endl;*/
}

//void rutine(float* arr, int n, int& position, float& p);

__device__ unsigned int block_id = -1;
__global__ void div(float* arr, int n, int position, float* p) {
	__shared__ float beg;
	beg = 0;
	if (threadIdx.x == 0) {
		beg = arr[(n)*(blockIdx.x + position) + position];
		/*if (beg == 3)
			return;*/
		/*__threadfence();*/
		
		int current_index = blockIdx.x;
		if (beg == 0 && current_index != 0)
			return;
		while (block_id != current_index - 1);
		*p *= beg;
		++block_id;
		/*__threadfence();*/
	}
	/*__syncthreads();*/
	if (beg == 0) {
		return;
	}
	unsigned int x = 1024*1024*(threadIdx.z) + 1024*(threadIdx.y) + (threadIdx.x) + position;
	unsigned int y = blockIdx.x + position;
	
	if ((x >= n) || (y >= n)) {
		return;
	}
	///*arr[y*n + x] = y + position;*/
	//if (x == 0) {
	//	beg = arr[(n)*y + position];
	//	if (beg == 0) {
	//		return;
	//	}
	//	*p *= beg;
	//}
	/*__syncthreads();*/
	/*if (beg != 0) {*/
		arr[y*(n)+x] /= beg;
	/*}*/


}
__global__ void sub(float* arr, int n, int position) {
	block_id = -1;
	__shared__ float beg;
	beg = 0;
	if (threadIdx.x == 0) {
		beg = arr[(n)*(blockIdx.x + position) + position];
	}
	/*unsigned int x = blockIdx.x + position;
	unsigned int y = 1024 * 1024 * (threadIdx.z) + 1024 * (threadIdx.y) + (threadIdx.x) + position;
	if (x >= n || y >= n) {
		return;
	}

	arr[y*(n)+x] -= arr[x + position*n];*/
	unsigned int x = 1024 * 1024 * (threadIdx.z) + 1024 * (threadIdx.y) + (threadIdx.x) + position;
	unsigned int y = blockIdx.x + position;
	__syncthreads();
	if ((x >= n) || (y >= n)) {
		return;
	}
	if (beg == 0) {
		return;
	}
	arr[y*(n)+x] -= arr[n*position + x];

}

int main() {
	SetConsoleCP(1251);
	SetConsoleOutputCP(1251);
	std::cout << "Введите размерность матрицы." << std::endl;
	int n;
	std::cin >> n;
	std::cout << "Введите элементы матрицы." << std::endl;
	srand(time(NULL));
	float *arr = new float[n*n];
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			std::cin >> arr[i*n + j];
		}
	}
	std::cout << std::endl;

	float* dev_arr = nullptr;
	float* dev_p = nullptr;
	float p = 1;
	int size = n*n;
	
	int position = 0;

	hipMalloc((void**)&dev_arr, size * sizeof(float));
	hipMalloc((void**)&dev_p, sizeof(float));

	hipMemcpy(dev_p, &p, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr, arr, size * sizeof(float), hipMemcpyHostToDevice);

	SYSTEMTIME s1, s2;
	GetLocalTime(&s1);
	while (position != n) {
		dim3 block(1 + (n - position) % 1024, 1/*1 + ((int)(n-position) / 1024 % 1024)*/, 1/*1 + (int)(n - position) / 1024 / 1024*/);
		dim3 grid((n - position), 1, 1);
		div<<<grid, block>>>(dev_arr, n, position, dev_p);
		/*hipMemcpy(arr, dev_arr, size * sizeof(float), hipMemcpyDeviceToHost);
		print(arr, n);
		hipMemcpy(&p, dev_p, sizeof(float), hipMemcpyDeviceToHost);
		std::cout << p << std::endl << std::endl;*/
		sub<<<grid, block>>>(dev_arr, n, position);
		/*hipMemcpy(arr, dev_arr, size * sizeof(float), hipMemcpyDeviceToHost);
		print(arr, n);
		hipMemcpy(&p, dev_p, sizeof(float), hipMemcpyDeviceToHost);
		std::cout << p << std::endl << std::endl;*/
		/*print(arr, n);
		rutine(arr, n, position, p);*/
		++position;
		/*hipMemcpy(dev_pos, &position, sizeof(float), hipMemcpyHostToDevice);*/
	}
	hipMemcpy(&p, dev_p, sizeof(float), hipMemcpyDeviceToHost);
	GetLocalTime(&s2);
	hipFree(dev_arr);
	hipFree(dev_p);
	
	std::cout << "Определитель: " << p << std::endl;
	std::cout << "Время: " << ((s2.wMinute - s1.wMinute) * 60 + s2.wSecond - s1.wSecond) * 1000
		+ s2.wMilliseconds - s1.wMilliseconds << std::endl;
	system("pause");
	return 0;
}

//void rutine(float* arr, int n, int& position) {
//	for (int i = position; i < n; ++i) {
//		float beg = arr[i*n + position];
//		p *= beg;
//		if (beg == 0) {
//			continue;
//		}
//		for (int j = position; j < n; ++j) {
//			arr[i*n + j] /= beg;
//		}
//	}
//
//	for (int i = position + 1; i < n; ++i) {
//		if (arr[i*n + position] == 0) {
//			continue;
//		}
//		for (int j = position; j < n; ++j) {
//			arr[i*n + j] -= arr[position*n + j];
//		}
//	}
//
//	++position;
//}