#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <string>
#include <Windows.h>


struct BMPHeader
{
	short type;
	int size;
	short reserved1;
	short reserved2;
	int offset;
};

struct BMPInfoHeader
{
	int size;
	int width;
	int height;
	short planes;
	short bitsPerPixel;
	unsigned compression;
	unsigned imageSize;
	int xPelsPerMeter;
	int yPelsPerMeter;
	int clrUsed;
	int clrImportant;
};

BMPHeader hdr1;
BMPInfoHeader infoHdr1;
bool first_load = true;
void LoadBMPFile(uchar4 **dst, int *width, int *height, const char *name)
{

	BMPHeader hdr;
	BMPInfoHeader infoHdr;
	int x, y;

	FILE *fd;


	printf("Loading %s...\n", name);

	if (sizeof(uchar4) != 4)
	{
		printf("***Bad uchar4 size***\n");
		exit(EXIT_SUCCESS);
	}

	if (!(fd = fopen(name, "rb")))
	{
		printf("***BMP load error: file access denied***\n");
		exit(EXIT_SUCCESS);
	}

	fread(&hdr.type, sizeof(hdr.type), 1, fd);
	fread(&hdr.size, sizeof(hdr.size), 1, fd);
	fread(&hdr.reserved1, sizeof(hdr.reserved1), 1, fd);
	fread(&hdr.reserved2, sizeof(hdr.reserved2), 1, fd);
	fread(&hdr.offset, sizeof(hdr.offset), 1, fd);

	if (hdr.type != 0x4D42)
	{
		printf("***BMP load error: bad file format***\n");
		exit(EXIT_SUCCESS);
	}

	fread(&infoHdr, sizeof(infoHdr), 1, fd);

	if (infoHdr.bitsPerPixel != 24)
	{
		printf("***BMP load error: invalid color depth***\n");
		exit(EXIT_SUCCESS);
	}

	if (infoHdr.compression)
	{
		printf("***BMP load error: compressed image***\n");
		exit(EXIT_SUCCESS);
	}
	if (first_load) {
		hdr1 = hdr;
		infoHdr1 = infoHdr;
		*width = infoHdr.width;
		*height = infoHdr.height;
		first_load = false;
	}
	*dst = (uchar4 *)malloc(infoHdr.width *infoHdr.height * 4);

	printf("BMP width: %u\n", infoHdr.width);
	printf("BMP height: %u\n", infoHdr.height);

	fseek(fd, hdr.offset, SEEK_SET);

	for (y = 0; y < infoHdr.height; y++)
	{
		for (x = 0; x < infoHdr.width; x++)
		{
			(*dst)[(y * infoHdr.width + x)].z = fgetc(fd);
			(*dst)[(y * infoHdr.width + x)].y = fgetc(fd);
			(*dst)[(y * infoHdr.width + x)].x = fgetc(fd);
		}

		for (x = 0; x < (4 - (3 * infoHdr.width) % 4) % 4; x++)
			fgetc(fd);
	}


	if (ferror(fd))
	{
		printf("***Unknown BMP load error.***\n");
		free(*dst);
		exit(EXIT_SUCCESS);
	}
	else
		printf("BMP file loaded successfully!\n");

	fclose(fd);
}

constexpr int block_size = 1024;
constexpr int grid_size = 1024;

__global__ void process(uchar4* bmp1, uchar4* bmp2, uchar4* bmp3, int width, int height) {
	__shared__ int count_x;
	__shared__ int count_y;
	count_x = 1;
	count_y = 1;
	if (threadIdx.x == 0) {
		count_x = (float)width / block_size + 1;
		count_y = (float)height / grid_size + 1;
	}

	__syncthreads();

	unsigned int x = threadIdx.x;
	unsigned int y = blockIdx.x;
	
	for (x = threadIdx.x;; --count_x, x += block_size) {
		if (x >= width) {
			return;
		}
		for (y = blockIdx.x;; --count_y, y += grid_size) {
			if (y >= height) {
				break;
			}
			int offset = y*width + x;
			float sum = bmp1[offset].x + bmp1[offset].y + bmp1[offset].z;
			sum /= 3;
			
			float mult1 = sum * bmp2[offset].x / 255;
			float mult2 = sum * bmp2[offset].y / 255;
			float mult3 = sum * bmp2[offset].z / 255;
		
			bmp3[offset].x = mult1;
			bmp3[offset].y = mult2;
			bmp3[offset].z = mult3;
			bmp3[offset].w = bmp1[offset].w;
		}
	}
}

uchar4* create_bmp(uchar4* bmp1, uchar4* bmp2, int &width, int &height) {
	uchar4* bmp = new uchar4[width*height];
	uchar4* dev_bmp1 = nullptr;
	uchar4* dev_bmp2 = nullptr;
	uchar4* dev_bmp3 = nullptr;
	
	int size = width*height;

	float* sum = new float[size];

	hipMalloc((void**)&dev_bmp1, size * sizeof(uchar4));
	hipMalloc((void**)&dev_bmp2, size * sizeof(uchar4));
	hipMalloc((void**)&dev_bmp3, size * sizeof(uchar4));

	hipMemcpy(dev_bmp1, bmp1, size * sizeof(uchar4), hipMemcpyHostToDevice);
	hipMemcpy(dev_bmp2, bmp2, size * sizeof(uchar4), hipMemcpyHostToDevice);

	process << <grid_size, block_size >> > (dev_bmp1, dev_bmp2, dev_bmp3, width, height);

	hipMemcpy(bmp, dev_bmp3, size * sizeof(uchar4), hipMemcpyDeviceToHost);
	
	hipFree(dev_bmp1);
	hipFree(dev_bmp2);
	hipFree(dev_bmp3);

	return bmp;
}

void SaveBMPFile(uchar4* uiData, const char *cName)
{
	int x, y;

	FILE *fd;

	fd = fopen(cName, "wb");

	fwrite(&hdr1.type, sizeof(hdr1.type), 1, fd);
	fwrite(&hdr1.size, sizeof(hdr1.size), 1, fd);
	fwrite(&hdr1.reserved1, sizeof(hdr1.reserved1), 1, fd);
	fwrite(&hdr1.reserved2, sizeof(hdr1.reserved2), 1, fd);
	fwrite(&hdr1.offset, sizeof(hdr1.offset), 1, fd);
	fwrite(&infoHdr1, sizeof(infoHdr1), 1, fd);
	
	fseek(fd, hdr1.offset, SEEK_SET);

	for (y = 0; y < infoHdr1.height; y++)
	{
		for (x = 0; x < infoHdr1.width; x++)
		{
			fputc(((uchar4*)uiData)[(y * infoHdr1.width + x)].z, fd);
			fputc(((uchar4*)uiData)[(y * infoHdr1.width + x)].y, fd);
			fputc(((uchar4*)uiData)[(y * infoHdr1.width + x)].x, fd);
		}

		for (x = 0; x < (4 - (3 * infoHdr1.width) % 4) % 4; x++)
		{
			fputc(' ', fd);
		}
	}
	printf("Image %s is saved.\n", cName);

	fclose(fd);
}

int main() {
	SetConsoleCP(1251);
	SetConsoleOutputCP(1251);

	uchar4* bmp1 = nullptr;
	uchar4* bmp2 = nullptr;
	uchar4* bmp3 = nullptr;
	int width = 0;
	int height = 0;
	while (true) {
		std::cout << "Все картинки должны быть в формате .bmp" << std::endl;
		std::cout << "1. Ввести адрес первой картинки." << std::endl
			<< "2. Ввести адрес второй картинки." << std::endl
			<< "3. Получить результирующую картинку." << std::endl
			<< "0. Завершение программы." << std::endl;

		int choose;
		std::cin >> choose;
		if (choose == 1) {
			std::cout << "Введите адрес первой картинки." << std::endl;
			std::string str;
			getchar();
			std::getline(std::cin, str);
			int* width1 = new int;
			int* height1 = new int;
			
			LoadBMPFile(&(bmp1), width1, height1, str.c_str());
			width = *width1;
			height = *height1;
		}
		else if (choose == 2) {
			std::cout << "Введите адрес второй картинки." << std::endl;
			std::string str;
			getchar();
			std::getline(std::cin, str);
			int* width1 = new int;
			int* height1 = new int;
			LoadBMPFile(&bmp2, width1, height1, str.c_str());
		}
		else if (choose == 3) {
			std::cout << "Введите адрес результирующей картинки." << std::endl;
			std::string str;
			getchar();
			std::getline(std::cin, str);
			bmp3 = create_bmp(bmp1, bmp2, width, height);
			SaveBMPFile(bmp3, str.c_str());
		}
		else if (choose == 0) {
			break;
		}
	}
	return 0;
}
